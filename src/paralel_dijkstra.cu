
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sys/time.h>

using namespace std;

#define N_MAX 9999

int* initializeGraf(int N, int* graf) {
	int r;
	int pembatas = 0;
	srand((unsigned)13517031);
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N - pembatas; j++) {
			r = rand() % 10;
			graf[i*N + j] = r;
			graf[j*N + i] = r;
		}
		pembatas += 1;
	}
	return graf;
}

void printOutput(int* matrix, int N, std::string filename) {
	ofstream file;
	file.open(filename, ios::out);

	if (file.is_open()) {
		for (int i = 0; i < N; i++) {
			file << "jarak dari node " << i << ": ";
			for (int j = 0; j < N; j++) {
				file << matrix[i*N + j] << " ";
			}
			file << endl;
		}
	}
	else {
		cout << "Error, unable to open file" << endl;
	}
	file.close();
}