
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sys/time.h>

using namespace std;

#define N_MAX 9999

int* initializeGraf(int N, int* graf) {
	int r;
	int pembatas = 0;
	srand((unsigned)13517031);
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N - pembatas; j++) {
			r = rand() % 10;
			graf[i*N + j] = r;
			graf[j*N + i] = r;
		}
		pembatas += 1;
	}
	return graf;
}